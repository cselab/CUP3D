#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#include "BiCGSTAB.cuh"

BiCGSTABSolver::BiCGSTABSolver(
    MPI_Comm m_comm,
    LocalSpMatDnVec& LocalLS,
    const int BLEN, 
    const int bMeanConstraint, 
    const std::vector<double>& P_inv)
  : m_comm_(m_comm), BLEN_(BLEN), bMeanConstraint_(bMeanConstraint), LocalLS_(LocalLS), prof_(m_comm)
{
  // MPI
  MPI_Comm_rank(m_comm_, &rank_);
  MPI_Comm_size(m_comm_, &comm_size_);

  // Set-up CUDA streams events, and handles
  checkCudaErrors(hipStreamCreate(&solver_stream_));
  checkCudaErrors(hipStreamCreate(&copy_stream_));
  checkCudaErrors(hipEventCreate(&sync_event_));
  checkCudaErrors(hipblasCreate(&cublas_handle_)); 
  checkCudaErrors(hipsparseCreate(&cusparse_handle_)); 
  // Set handles to stream
  checkCudaErrors(hipblasSetStream(cublas_handle_, solver_stream_));
  checkCudaErrors(hipsparseSetStream(cusparse_handle_, solver_stream_));
  // Set pointer modes to device
  checkCudaErrors(hipblasSetPointerMode(cublas_handle_, HIPBLAS_POINTER_MODE_DEVICE));
  checkCudaErrors(hipsparseSetPointerMode(cusparse_handle_, HIPSPARSE_POINTER_MODE_DEVICE));

  // Set constants and allocate memory for scalars
  double h_consts[3] = {1., -1., 0.};
  checkCudaErrors(hipMalloc(&d_consts_, 3 * sizeof(double)));
  checkCudaErrors(hipMemcpyAsync(d_consts_, h_consts, 3 * sizeof(double), hipMemcpyHostToDevice, solver_stream_));
  d_eye_ = d_consts_;
  d_nye_ = d_consts_ + 1;
  d_nil_ = d_consts_ + 2;
  checkCudaErrors(hipMalloc(&d_coeffs_, sizeof(BiCGSTABScalars)));
  checkCudaErrors(hipHostMalloc(&h_coeffs_, sizeof(BiCGSTABScalars)));

  // Copy preconditionner
  checkCudaErrors(hipMalloc(&d_P_inv_, BLEN_ * BLEN_ * sizeof(double)));
  checkCudaErrors(hipMemcpyAsync(d_P_inv_, P_inv.data(), BLEN_ * BLEN_ * sizeof(double), hipMemcpyHostToDevice, solver_stream_));

}

BiCGSTABSolver::~BiCGSTABSolver()
{
  // Cleanup after last timestep
  this->freeLast();

  prof_.print("Total");

  // Free preconditionner
  checkCudaErrors(hipFree(d_P_inv_));

  // Free device consants
  checkCudaErrors(hipFree(d_consts_));
  checkCudaErrors(hipFree(d_coeffs_));
  checkCudaErrors(hipHostFree(h_coeffs_));

  // Destroy CUDA streams and handles
  checkCudaErrors(hipblasDestroy(cublas_handle_)); 
  checkCudaErrors(hipsparseDestroy(cusparse_handle_)); 
  checkCudaErrors(hipEventDestroy(sync_event_));
  checkCudaErrors(hipStreamDestroy(copy_stream_));
  checkCudaErrors(hipStreamDestroy(solver_stream_));
}

// --------------------------------- public class methods ------------------------------------

void BiCGSTABSolver::solveWithUpdate(
    const double max_error,
    const double max_rel_error,
    const int max_restarts)
{

  this->updateAll();
  this->main(max_error, max_rel_error, max_restarts);
}

void BiCGSTABSolver::solveNoUpdate(
    const double max_error,
    const double max_rel_error,
    const int max_restarts)
{
  this->updateVec();
  this->main(max_error, max_rel_error, max_restarts);
}

// --------------------------------- private class methods ------------------------------------

void BiCGSTABSolver::freeLast()
{
  if (dirty_) // Previous time-step exists so cleanup first
  {
    // Free device memory allocated for linear system from previous time-step
    checkCudaErrors(hipFree(dloc_cooValA_));
    checkCudaErrors(hipFree(dloc_cooRowA_));
    checkCudaErrors(hipFree(dloc_cooColA_));
    checkCudaErrors(hipFree(d_x_)); 
    checkCudaErrors(hipFree(d_x_opt_)); 
    checkCudaErrors(hipFree(d_r_));
    checkCudaErrors(hipFree(d_h3_));
    checkCudaErrors(hipFree(d_invh_));
    checkCudaErrors(hipFree(d_red_));
    checkCudaErrors(hipFree(d_red_res_));
    // Cleanup memory allocated for BiCGSTAB arrays
    checkCudaErrors(hipFree(d_rhat_));
    checkCudaErrors(hipFree(d_p_));
    checkCudaErrors(hipFree(d_nu_));
    checkCudaErrors(hipFree(d_t_));
    checkCudaErrors(hipFree(d_z_));
    // Free and destroy cuSPARSE memory/descriptors
    checkCudaErrors(hipFree(locSpMVBuff_));
    checkCudaErrors(hipsparseDestroySpMat(spDescrLocA_));
    checkCudaErrors(hipsparseDestroyDnVec(spDescrNu_));
    checkCudaErrors(hipsparseDestroyDnVec(spDescrT_));
    checkCudaErrors(hipsparseDestroyDnVec(spDescrLocZ_));
    if (comm_size_ > 1)
    {
      checkCudaErrors(hipFree(d_send_pack_idx_));
      checkCudaErrors(hipFree(d_send_buff_));
      checkCudaErrors(hipHostFree(h_send_buff_));
      checkCudaErrors(hipHostFree(h_recv_buff_));
      checkCudaErrors(hipFree(dbd_cooValA_));
      checkCudaErrors(hipFree(dbd_cooRowA_));
      checkCudaErrors(hipFree(dbd_cooColA_));
      checkCudaErrors(hipFree(bdSpMVBuff_));
      checkCudaErrors(hipsparseDestroySpMat(spDescrBdA_));
      checkCudaErrors(hipsparseDestroyDnVec(spDescrBdZ_));
    }
  }
  dirty_ = true;
}

void BiCGSTABSolver::updateAll()
{
  this->freeLast();

  // Update LS metadata
  m_ = LocalLS_.m_;
  halo_ = LocalLS_.halo_ ;
  hd_m_ = m_ + halo_;
  loc_nnz_ = LocalLS_.loc_nnz_ ;
  bd_nnz_ = LocalLS_.bd_nnz_ ;
  send_buff_sz_ = LocalLS_.send_pack_idx_.size();
  const int Nblocks = m_ / BLEN_;
  
  // Allocate device memory for local linear system
  checkCudaErrors(hipMalloc(&dloc_cooValA_, loc_nnz_ * sizeof(double)));
  checkCudaErrors(hipMalloc(&dloc_cooRowA_, loc_nnz_ * sizeof(int)));
  checkCudaErrors(hipMalloc(&dloc_cooColA_, loc_nnz_ * sizeof(int)));
  checkCudaErrors(hipMalloc(&d_x_, m_ * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_x_opt_, m_ * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_r_, m_ * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_h3_, Nblocks * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_invh_, Nblocks * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_red_, m_ * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_red_res_, sizeof(double)));
  // Allocate arrays for BiCGSTAB storage
  checkCudaErrors(hipMalloc(&d_rhat_, m_ * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_p_, m_ * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_nu_, m_ * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_t_,  m_ * sizeof(double)));
  checkCudaErrors(hipMalloc(&d_z_,  hd_m_ * sizeof(double)));
  if (comm_size_ > 1)
  {
    checkCudaErrors(hipMalloc(&d_send_pack_idx_, send_buff_sz_ * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_send_buff_, send_buff_sz_ * sizeof(double)));
    checkCudaErrors(hipHostMalloc(&h_send_buff_, send_buff_sz_ * sizeof(double)));
    checkCudaErrors(hipHostMalloc(&h_recv_buff_, halo_ * sizeof(double)));
    checkCudaErrors(hipMalloc(&dbd_cooValA_, bd_nnz_ * sizeof(double)));
    checkCudaErrors(hipMalloc(&dbd_cooRowA_, bd_nnz_ * sizeof(int)));
    checkCudaErrors(hipMalloc(&dbd_cooColA_, bd_nnz_ * sizeof(int)));
  }

  prof_.startProfiler("Memcpy", solver_stream_);
  // H2D transfer of linear system
  checkCudaErrors(hipMemcpyAsync(dloc_cooValA_, LocalLS_.loc_cooValA_.data(), loc_nnz_ * sizeof(double), hipMemcpyHostToDevice, solver_stream_));
  checkCudaErrors(hipMemcpyAsync(dloc_cooRowA_, LocalLS_.loc_cooRowA_int_.data(), loc_nnz_ * sizeof(int), hipMemcpyHostToDevice, solver_stream_));
  checkCudaErrors(hipMemcpyAsync(dloc_cooColA_, LocalLS_.loc_cooColA_int_.data(), loc_nnz_ * sizeof(int), hipMemcpyHostToDevice, solver_stream_));
  checkCudaErrors(hipMemcpyAsync(d_h3_, LocalLS_.h3_.data(), Nblocks * sizeof(double), hipMemcpyHostToDevice, solver_stream_));
  checkCudaErrors(hipMemcpyAsync(d_invh_, LocalLS_.invh_.data(), Nblocks * sizeof(double), hipMemcpyHostToDevice, solver_stream_));
  if (comm_size_ > 1)
  {
    checkCudaErrors(hipMemcpyAsync(d_send_pack_idx_, LocalLS_.send_pack_idx_.data(), send_buff_sz_ * sizeof(int), hipMemcpyHostToDevice, solver_stream_));
    checkCudaErrors(hipMemcpyAsync(dbd_cooValA_, LocalLS_.bd_cooValA_.data(), bd_nnz_ * sizeof(double), hipMemcpyHostToDevice, solver_stream_));
    checkCudaErrors(hipMemcpyAsync(dbd_cooRowA_, LocalLS_.bd_cooRowA_int_.data(), bd_nnz_ * sizeof(int), hipMemcpyHostToDevice, solver_stream_));
    checkCudaErrors(hipMemcpyAsync(dbd_cooColA_, LocalLS_.bd_cooColA_int_.data(), bd_nnz_ * sizeof(int), hipMemcpyHostToDevice, solver_stream_));
  }
  prof_.stopProfiler("Memcpy", solver_stream_);

  // Create descriptors for variables that will pass through cuSPARSE
  checkCudaErrors(hipsparseCreateCoo(&spDescrLocA_, m_, m_, loc_nnz_, dloc_cooRowA_, dloc_cooColA_, dloc_cooValA_, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
  checkCudaErrors(hipsparseCreateDnVec(&spDescrNu_, m_, d_nu_, HIP_R_64F));
  checkCudaErrors(hipsparseCreateDnVec(&spDescrT_, m_, d_t_, HIP_R_64F));
  checkCudaErrors(hipsparseCreateDnVec(&spDescrLocZ_, m_, d_z_, HIP_R_64F));
  // Allocate work buffer for hipsparseSpMV
  checkCudaErrors(hipsparseSpMV_bufferSize(
        cusparse_handle_, 
        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        d_eye_, 
        spDescrLocA_, 
        spDescrLocZ_, 
        d_nil_, 
        spDescrNu_, 
        HIP_R_64F, 
        HIPSPARSE_MV_ALG_DEFAULT, 
        &locSpMVBuffSz_));
  checkCudaErrors(hipMalloc(&locSpMVBuff_, locSpMVBuffSz_ * sizeof(char)));
  if (comm_size_ > 1)
  {
    checkCudaErrors(hipsparseCreateCoo(&spDescrBdA_, m_, hd_m_, bd_nnz_, dbd_cooRowA_, dbd_cooColA_, dbd_cooValA_, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&spDescrBdZ_, hd_m_, d_z_, HIP_R_64F));
    checkCudaErrors(hipsparseSpMV_bufferSize(
          cusparse_handle_, 
          HIPSPARSE_OPERATION_NON_TRANSPOSE, 
          d_eye_, 
          spDescrBdA_, 
          spDescrBdZ_, 
          d_eye_, 
          spDescrNu_, 
          HIP_R_64F, 
          HIPSPARSE_MV_ALG_DEFAULT, 
          &bdSpMVBuffSz_));
    checkCudaErrors(hipMalloc(&bdSpMVBuff_, bdSpMVBuffSz_ * sizeof(char)));
  }

  this->updateVec();
}

void BiCGSTABSolver::updateVec()
{
  prof_.startProfiler("Memcpy", solver_stream_);
  // Copy RHS, LHS vec initial guess (to d_z_), if LS was updated, updateAll reallocates sufficient memory
  checkCudaErrors(hipMemcpyAsync(d_x_, LocalLS_.x_.data(), m_ * sizeof(double), hipMemcpyHostToDevice, solver_stream_));
  checkCudaErrors(hipMemcpyAsync(d_r_, LocalLS_.b_.data(), m_ * sizeof(double), hipMemcpyHostToDevice, solver_stream_));
  prof_.stopProfiler("Memcpy", solver_stream_);
}

__global__ void set_negative(double* const dest, double* const source)
{
  dest[0] = -source[0];
}

__global__ void breakdown_update(BiCGSTABScalars* coeffs)
{
  coeffs->rho_prev = 1.;
  coeffs->alpha = 1.;
  coeffs->omega = 1.;
  coeffs->beta = (coeffs->rho_curr / (coeffs->rho_prev + coeffs->eps)) * (coeffs->alpha / (coeffs->omega + coeffs->eps));
}

__global__ void set_beta(BiCGSTABScalars* coeffs)
{
  coeffs->beta = (coeffs->rho_curr / (coeffs->rho_prev + coeffs->eps)) * (coeffs->alpha / (coeffs->omega + coeffs->eps));
}

__global__ void set_alpha(BiCGSTABScalars* coeffs)
{
  coeffs->alpha = coeffs->rho_curr / (coeffs->buff_1 + coeffs->eps);
}

__global__ void set_omega(BiCGSTABScalars* coeffs)
{
  coeffs->omega = coeffs->buff_1 / (coeffs->buff_2 + coeffs->eps);
}

__global__ void set_rho(BiCGSTABScalars* coeffs)
{
  coeffs->rho_prev = coeffs->rho_curr;
}

__global__ void blockDscal(const int m, const int BLEN, const double* __restrict__ const alpha, double* __restrict__ const x)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < m; i += blockDim.x * gridDim.x)
    x[i] = alpha[i/BLEN] * x[i];

}

__global__ void send_buff_pack(
    int buff_sz, 
    const int* const pack_idx, 
    double* __restrict__ const buff, 
    const double* __restrict__ const source)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < buff_sz; i += blockDim.x * gridDim.x)
    buff[i] = source[pack_idx[i]];
}

__global__ void bMean2Apply(const int m, const int BLEN, const double red_res, const double* __restrict__ const h3, double* const x)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < m; i += blockDim.x * gridDim.x)
    x[i] = x[i] + m*h3[i/BLEN];
}

void BiCGSTABSolver::hd_cusparseSpMV(
  double* d_op_hd,  // operand vec
  hipsparseDnVecDescr_t spDescrLocOp,
  hipsparseDnVecDescr_t spDescrBdOp,
  double* d_res_hd, // result vec
  hipsparseDnVecDescr_t spDescrRes)
{

  const std::vector<int> &recv_ranks = LocalLS_.recv_ranks_;
  const std::vector<int> &recv_offset = LocalLS_.recv_offset_;
  const std::vector<int> &recv_sz = LocalLS_.recv_sz_;
  const std::vector<int> &send_ranks = LocalLS_.send_ranks_;
  const std::vector<int> &send_offset = LocalLS_.send_offset_;
  const std::vector<int> &send_sz = LocalLS_.send_sz_;

  if (comm_size_ > 1)
  {
    send_buff_pack<<<8*56,32, 0, solver_stream_>>>(send_buff_sz_, d_send_pack_idx_, d_send_buff_, d_op_hd);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipEventRecord(sync_event_, solver_stream_)); // event to sync up for MPI comm

  }

  prof_.startProfiler("KerSpMV", solver_stream_);
  // A*x for local rows
  checkCudaErrors(hipsparseSpMV( 
        cusparse_handle_, 
        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        d_eye_, 
        spDescrLocA_, 
        spDescrLocOp, 
        d_nil_, 
        spDescrRes, 
        HIP_R_64F, 
        HIPSPARSE_MV_ALG_DEFAULT, 
        locSpMVBuff_)); 
  prof_.stopProfiler("KerSpMV", solver_stream_);

  if (comm_size_ > 1)
  {
    prof_.startProfiler("HaloComm", copy_stream_);
    // Wait until copy to buffer has completed
    checkCudaErrors(hipStreamWaitEvent(copy_stream_, sync_event_, 0));
    checkCudaErrors(hipMemcpyAsync(h_send_buff_, d_send_buff_, send_buff_sz_ * sizeof(double), hipMemcpyDeviceToHost, copy_stream_));
    checkCudaErrors(hipStreamSynchronize(copy_stream_));

    // Schedule receives and wait for them to arrive
    std::vector<MPI_Request> recv_requests(recv_ranks.size());
    for (size_t i(0); i < recv_ranks.size(); i++)
      MPI_Irecv(&h_recv_buff_[recv_offset[i]], recv_sz[i], MPI_DOUBLE, recv_ranks[i], 978, m_comm_, &recv_requests[i]);

    std::vector<MPI_Request> send_requests(send_ranks.size());
    for (size_t i(0); i < send_ranks.size(); i++)
      MPI_Isend(&h_send_buff_[send_offset[i]], send_sz[i], MPI_DOUBLE, send_ranks[i], 978, m_comm_, &send_requests[i]);

    MPI_Waitall(send_ranks.size(), send_requests.data(), MPI_STATUS_IGNORE);
    MPI_Waitall(recv_ranks.size(), recv_requests.data(), MPI_STATUS_IGNORE);
    prof_.stopProfiler("HaloComm", copy_stream_);

    // Use solver stream, just in case... even though the halo doesn't particiapte in SpMV race conditions possible due to coalescing?
    checkCudaErrors(hipMemcpyAsync(&d_op_hd[m_], h_recv_buff_, halo_ * sizeof(double), hipMemcpyHostToDevice, solver_stream_));

    prof_.startProfiler("HaloSpMV", solver_stream_);
    // A*x for rows with halo elements, axpy with local results
    checkCudaErrors(hipsparseSpMV( 
          cusparse_handle_, 
          HIPSPARSE_OPERATION_NON_TRANSPOSE, 
          d_eye_, 
          spDescrBdA_, 
          spDescrBdOp, 
          d_eye_, 
          spDescrRes, 
          HIP_R_64F, 
          HIPSPARSE_MV_ALG_DEFAULT, 
          bdSpMVBuff_)); 
    prof_.stopProfiler("HaloSpMV", solver_stream_);
  }

  if (bMeanConstraint_ == 1 || bMeanConstraint_ == 2)
  {
    // Copy result to reduction buffer and scale by h_i^3
    checkCudaErrors(hipMemcpyAsync(d_red_, d_res_hd, m_ * sizeof(double), hipMemcpyDeviceToDevice, solver_stream_));
    blockDscal<<<8*56, 128, 0, solver_stream_>>>(m_, BLEN_, d_h3_, d_red_);
    checkCudaErrors(hipGetLastError());

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum<double*, double*>(d_temp_storage, temp_storage_bytes, d_red_, d_red_res_, m_, solver_stream_);

    double h_red_res;
    checkCudaErrors(hipMemcpyAsync(&h_red_res, d_red_res_, sizeof(double), hipMemcpyDeviceToHost, solver_stream_));
    checkCudaErrors(hipStreamSynchronize(solver_stream_));
    MPI_Allreduce(MPI_IN_PLACE, &h_red_res, 1, MPI_DOUBLE, MPI_SUM, m_comm_);
    
    if (bMeanConstraint_ == 1 && bMeanRow_ >= 0)
      checkCudaErrors(hipMemcpyAsync(&d_res_hd[bMeanRow_], &h_red_res, sizeof(double), hipMemcpyHostToDevice, solver_stream_));
    else if (bMeanConstraint_ == 2)
    {
      bMean2Apply<<<8*56, 128, 0, solver_stream_>>>(m_, BLEN_, h_red_res, d_h3_, d_res_hd); 
      checkCudaErrors(hipGetLastError());
    }
  }
}

void BiCGSTABSolver::main(
    const double max_error, 
    const double max_rel_error, 
    const int max_restarts)
{
  prof_.startProfiler("Total", solver_stream_);

  // Initialize variables to evaluate convergence
  double error = 1e50;
  double error_init = 1e50;
  double error_opt = 1e50;
  bool bConverged = false;
  int restarts = 0;

  // 3. Set initial values to scalars
  *h_coeffs_ = {1., 1., 1., 1e-21, 1., 1., 0., 0.};
  checkCudaErrors(hipMemcpyAsync(d_coeffs_, h_coeffs_, sizeof(BiCGSTABScalars), hipMemcpyHostToDevice, solver_stream_));

  // 1. r <- b - A*x_0.  Add bias with cuBLAS like in "NVIDIA_CUDA-11.4_Samples/7_CUDALibraries/conjugateGradient"
  checkCudaErrors(hipMemcpyAsync(d_z_, d_x_, m_ * sizeof(double), hipMemcpyDeviceToDevice, solver_stream_));
	hd_cusparseSpMV(d_z_, spDescrLocZ_, spDescrBdZ_, d_nu_, spDescrNu_);
  checkCudaErrors(hipblasDaxpy(cublas_handle_, m_, d_nye_, d_nu_, 1, d_r_, 1)); // r <- -A*x_0 + b

  // ||A*x_0||
  checkCudaErrors(hipblasDnrm2(cublas_handle_, m_, d_nu_, 1, &(d_coeffs_->buff_1)));
  checkCudaErrors(hipGetLastError());
  // ||b - A*x_0||
  checkCudaErrors(hipblasDnrm2(cublas_handle_, m_, d_r_, 1, &(d_coeffs_->buff_2)));
  checkCudaErrors(hipGetLastError());
  // buff_1 and buff_2 in contigious memory in BiCGSTABScalars
  checkCudaErrors(hipMemcpyAsync(&(h_coeffs_->buff_1), &(d_coeffs_->buff_1), 2*sizeof(double), hipMemcpyDeviceToHost, solver_stream_));
  checkCudaErrors(hipStreamSynchronize(solver_stream_));

  h_coeffs_->buff_1 *= h_coeffs_->buff_1; // get square norm
  h_coeffs_->buff_2 *= h_coeffs_->buff_2;
  MPI_Allreduce(MPI_IN_PLACE, &(h_coeffs_->buff_1), 2, MPI_DOUBLE, MPI_SUM, m_comm_);
  h_coeffs_->buff_1 = std::sqrt(h_coeffs_->buff_1);
  h_coeffs_->buff_2 = std::sqrt(h_coeffs_->buff_2);

  if (rank_ == 0)
  {
    std::cout << "  [BiCGSTAB]: || A*x_0 || = " << h_coeffs_->buff_1 << std::endl;
    std::cout << "  [BiCGSTAB]: Initial norm: " << h_coeffs_->buff_2 << std::endl;
  }
  // Set initial error and x_opt
  error = h_coeffs_->buff_2;
  error_init = error;
  error_opt = error;
  checkCudaErrors(hipMemcpyAsync(d_x_opt_, d_x_, m_ * sizeof(double), hipMemcpyDeviceToDevice, solver_stream_));

  // 2. Set r_hat = r
  checkCudaErrors(hipMemcpyAsync(d_rhat_, d_r_, m_ * sizeof(double), hipMemcpyDeviceToDevice, solver_stream_));

  // 4. Set initial values of vectors to zero
  checkCudaErrors(hipMemsetAsync(d_nu_, 0, m_ * sizeof(double), solver_stream_));
  checkCudaErrors(hipMemsetAsync(d_p_, 0, m_ * sizeof(double), solver_stream_));

  // 5. Start iterations
  const size_t max_iter = 1000;
  for(size_t k(0); k<max_iter; k++)
  {
    // 1. rho_i = (r_hat, r)
    checkCudaErrors(hipblasDdot(cublas_handle_, m_, d_rhat_, 1, d_r_, 1, &(d_coeffs_->rho_curr)));
    
    // Numerical convergence trick
    checkCudaErrors(hipblasDnrm2(cublas_handle_, m_, d_r_, 1, &(d_coeffs_->buff_1)));
    checkCudaErrors(hipblasDnrm2(cublas_handle_, m_, d_rhat_, 1, &(d_coeffs_->buff_2)));
    checkCudaErrors(hipMemcpyAsync(&(h_coeffs_->rho_curr), &(d_coeffs_->rho_curr), 3 * sizeof(double), hipMemcpyDeviceToHost, solver_stream_));
    checkCudaErrors(hipStreamSynchronize(solver_stream_)); 
    h_coeffs_->buff_1 *= h_coeffs_->buff_1; // get square norm
    h_coeffs_->buff_2 *= h_coeffs_->buff_2;
    MPI_Allreduce(MPI_IN_PLACE, &(h_coeffs_->rho_curr), 3, MPI_DOUBLE, MPI_SUM, m_comm_);
    checkCudaErrors(hipMemcpyAsync(&(d_coeffs_->rho_curr), &(h_coeffs_->rho_curr), sizeof(double), hipMemcpyHostToDevice, solver_stream_));
    const bool serious_breakdown = h_coeffs_->rho_curr * h_coeffs_->rho_curr < 1e-16 * h_coeffs_->buff_1 * h_coeffs_->buff_2;

    // 2. beta = (rho_i / rho_{i-1}) * (alpha / omega_{i-1})
    set_beta<<<1, 1, 0, solver_stream_>>>(d_coeffs_);
    checkCudaErrors(hipGetLastError());
    if(serious_breakdown && max_restarts > 0)
    {
      restarts++;
      if(restarts >= max_restarts){
        break;
      }
      if (rank_ == 0)
      {
        std::cout << "  [BiCGSTAB]: Restart at iteration: " << k << " norm: " << error <<" Initial norm: " << error_init << std::endl;
      }
      checkCudaErrors(hipMemcpyAsync(d_rhat_, d_r_, m_ * sizeof(double), hipMemcpyDeviceToDevice, solver_stream_));
      checkCudaErrors(hipblasDnrm2(cublas_handle_, m_, d_rhat_, 1, &(d_coeffs_->rho_curr)));
      checkCudaErrors(hipMemcpyAsync(&(h_coeffs_->rho_curr), &(d_coeffs_->rho_curr), sizeof(double), hipMemcpyDeviceToHost, solver_stream_));
      checkCudaErrors(hipStreamSynchronize(solver_stream_));
      h_coeffs_->rho_curr *= h_coeffs_->rho_curr;
      MPI_Allreduce(MPI_IN_PLACE, &(h_coeffs_->rho_curr), 1, MPI_DOUBLE, MPI_SUM, m_comm_);
      checkCudaErrors(hipMemcpyAsync(&(d_coeffs_->rho_curr), &(h_coeffs_->rho_curr), sizeof(double), hipMemcpyHostToDevice, solver_stream_));
      checkCudaErrors(hipMemsetAsync(d_nu_, 0, m_ * sizeof(double), solver_stream_));
      checkCudaErrors(hipMemsetAsync(d_p_, 0, m_ * sizeof(double), solver_stream_));
      breakdown_update<<<1, 1, 0, solver_stream_>>>(d_coeffs_);
      checkCudaErrors(hipGetLastError());
    }

    // 3. p_i = r_{i-1} + beta(p_{i-1} - omega_{i-1}*nu_i)
    set_negative<<<1, 1, 0, solver_stream_>>>(&(d_coeffs_->buff_1), &(d_coeffs_->omega));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipblasDaxpy(cublas_handle_, m_, &(d_coeffs_->buff_1), d_nu_, 1, d_p_, 1)); // p <- -omega_{i-1}*nu_i + p
    checkCudaErrors(hipblasDscal(cublas_handle_, m_, &(d_coeffs_->beta), d_p_, 1));            // p <- beta * p
    checkCudaErrors(hipblasDaxpy(cublas_handle_, m_, d_eye_, d_r_, 1, d_p_, 1));    // p <- r_{i-1} + p

    // 4. z <- K_2^{-1} * p_i
    prof_.startProfiler("Prec", solver_stream_);
    checkCudaErrors(hipblasDgemm(cublas_handle_, HIPBLAS_OP_T, HIPBLAS_OP_N, BLEN_, m_ / BLEN_, BLEN_, d_eye_, d_P_inv_, BLEN_, d_p_, BLEN_, d_nil_, d_z_, BLEN_));
    blockDscal<<<8*56, 128, 0, solver_stream_>>>(m_, BLEN_, d_invh_, d_z_);
    checkCudaErrors(hipGetLastError());
    prof_.stopProfiler("Prec", solver_stream_);

    // 5. nu_i = A * z
	  hd_cusparseSpMV(d_z_, spDescrLocZ_, spDescrBdZ_, d_nu_, spDescrNu_);

    // 6. alpha = rho_i / (r_hat, nu_i)
    checkCudaErrors(hipblasDdot(cublas_handle_, m_, d_rhat_, 1, d_nu_, 1, &(d_coeffs_->buff_1)));
    checkCudaErrors(hipMemcpyAsync(&(h_coeffs_->buff_1), &(d_coeffs_->buff_1), sizeof(double), hipMemcpyDeviceToHost, solver_stream_));
    checkCudaErrors(hipStreamSynchronize(solver_stream_));
    MPI_Allreduce(MPI_IN_PLACE, &(h_coeffs_->buff_1), 1, MPI_DOUBLE, MPI_SUM, m_comm_);
    checkCudaErrors(hipMemcpyAsync(&(d_coeffs_->buff_1), &(h_coeffs_->buff_1), sizeof(double), hipMemcpyHostToDevice, solver_stream_));
    set_alpha<<<1, 1, 0, solver_stream_>>>(d_coeffs_);
    checkCudaErrors(hipGetLastError());

    // 7. h = alpha*z + x_{i-1}
    checkCudaErrors(hipblasDaxpy(cublas_handle_, m_, &(d_coeffs_->alpha), d_z_, 1, d_x_, 1));

    // 9. s = -alpha * nu_i + r_{i-1}
    set_negative<<<1, 1, 0, solver_stream_>>>(&(d_coeffs_->buff_1), &(d_coeffs_->alpha));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipblasDaxpy(cublas_handle_, m_, &(d_coeffs_->buff_1), d_nu_, 1, d_r_, 1));

    // 10. z <- K_2^{-1} * s
    prof_.startProfiler("Prec", solver_stream_);
    checkCudaErrors(hipblasDgemm(cublas_handle_, HIPBLAS_OP_T, HIPBLAS_OP_N, BLEN_, m_ / BLEN_, BLEN_, d_eye_, d_P_inv_, BLEN_, d_r_, BLEN_, d_nil_, d_z_, BLEN_));
    blockDscal<<<8*56, 128, 0, solver_stream_>>>(m_, BLEN_, d_invh_, d_z_);
    checkCudaErrors(hipGetLastError());
    prof_.stopProfiler("Prec", solver_stream_);

    // 11. t = A * z
	  hd_cusparseSpMV(d_z_, spDescrLocZ_, spDescrBdZ_, d_t_, spDescrT_);
    
    // 12. omega_i = (t,s)/(t,t), variables alpha & beta no longer in use this iter
    checkCudaErrors(hipblasDdot(cublas_handle_, m_, d_t_, 1, d_r_, 1, &(d_coeffs_->buff_1)));
    checkCudaErrors(hipblasDnrm2(cublas_handle_, m_, d_t_, 1, &(d_coeffs_->buff_2)));
    checkCudaErrors(hipMemcpyAsync(&(h_coeffs_->buff_1), &(d_coeffs_->buff_1), 2 * sizeof(double), hipMemcpyDeviceToHost, solver_stream_));
    checkCudaErrors(hipStreamSynchronize(solver_stream_));
    h_coeffs_->buff_2 *= h_coeffs_->buff_2;
    MPI_Allreduce(MPI_IN_PLACE, &(h_coeffs_->buff_1), 2, MPI_DOUBLE, MPI_SUM, m_comm_);
    checkCudaErrors(hipMemcpyAsync(&(d_coeffs_->buff_1), &(h_coeffs_->buff_1), 2 * sizeof(double), hipMemcpyHostToDevice, solver_stream_));
    set_omega<<<1, 1, 0, solver_stream_>>>(d_coeffs_);
    checkCudaErrors(hipGetLastError());

    // 13. x_i = omega_i * z + h
    checkCudaErrors(hipblasDaxpy(cublas_handle_, m_, &(d_coeffs_->omega), d_z_, 1, d_x_, 1));

    // 15. r_i = -omega_i * t + s
    set_negative<<<1, 1, 0, solver_stream_>>>(&(d_coeffs_->buff_1), &(d_coeffs_->omega));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipblasDaxpy(cublas_handle_, m_, &(d_coeffs_->buff_1), d_t_, 1, d_r_, 1));

    // If x_i accurate enough then quit
    checkCudaErrors(hipblasDnrm2(cublas_handle_, m_, d_r_, 1, &(d_coeffs_->buff_1)));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpyAsync(&error, &(d_coeffs_->buff_1), sizeof(double), hipMemcpyDeviceToHost, solver_stream_));
    checkCudaErrors(hipStreamSynchronize(solver_stream_));

    error *= error;
    MPI_Allreduce(MPI_IN_PLACE, &error, 1, MPI_DOUBLE, MPI_SUM, m_comm_);
    error = std::sqrt(error);

    if (error < error_opt)
    {
      error_opt = error;
      checkCudaErrors(hipMemcpyAsync(d_x_opt_, d_x_, m_ * sizeof(double), hipMemcpyDeviceToDevice, solver_stream_));

      if((error <= max_error) || (error / error_init <= max_rel_error))
      {
        if (rank_ == 0)
          std::cout << "  [BiCGSTAB]: Converged after " << k << " iterations" << std::endl;;

        bConverged = true;
        break;
      }
    }


    // Update *_prev values for next iteration
    set_rho<<<1, 1, 0, solver_stream_>>>(d_coeffs_);
    checkCudaErrors(hipGetLastError());

  }

  if (rank_ == 0)
  {
    if( bConverged )
      std::cout <<  "  [BiCGSTAB] Error norm (relative) = " << error_opt << "/" << max_error 
                << " (" << error_opt/error_init  << "/" << max_rel_error << ")" << std::endl;
    else
      std::cout <<  "  [BiCGSTAB]: Iteration " << max_iter 
                << ". Error norm (relative) = " << error_opt << "/" << max_error 
                << " (" << error_opt/error_init  << "/" << max_rel_error << ")" << std::endl;
  }

  prof_.startProfiler("Memcpy", solver_stream_);
  // Copy result back to host
  checkCudaErrors(hipMemcpyAsync(LocalLS_.x_.data(), d_x_opt_, m_ * sizeof(double), hipMemcpyDeviceToHost, solver_stream_));
  prof_.stopProfiler("Memcpy", solver_stream_);
  prof_.stopProfiler("Total", solver_stream_);
}

